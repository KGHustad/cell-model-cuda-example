#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>

int main()
{
    int device_count = 0;
    hipError_t error_id = hipGetDeviceCount(&device_count);

    if (error_id != hipSuccess) {
        printf("hipGetDeviceCount returned %d\n-> %s\n",
               (int) (error_id), hipGetErrorString(error_id));
        printf("Result = FAIL\n");
        exit(EXIT_FAILURE);
    }

    for (int device = 0; device < device_count; device++) {
        hipSetDevice(device);
        struct hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, device);

        printf("%d%d", deviceProp.major, deviceProp.minor);
        if (device < device_count-1) {
            printf(";");
        }
    }

    return 0;
}
